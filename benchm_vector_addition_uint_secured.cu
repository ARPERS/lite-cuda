#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <typeinfo>

#include <chrono>

#include "lite.cu"

__global__ void vectorAdditionUnsecure(uint *result, uint *a, uint *b, int N){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for(int i = index; i < N; i += stride){
        result[i] = a[i] + b[i];
    }
}

void check(uint *a, uint *b, uint *array, int N){
    bool flag = false;
    for(int i = 0; i < N; i++){
        if(array[i] != a[i]+b[i]){
            flag = true;
        }
    }
    if(!flag){
        printf("SUCCESS\n");
        // for(int i = 0; i < N; i++) printf("%u ", a[i]); printf("\n");
        // for(int i = 0; i < N; i++) printf("%u ", b[i]); printf("\n");
        // for(int i = 0; i < N; i++) printf("%u ", array[i]); printf("\n");
    }else{
        printf("FAIL\n");
        // for(int i = 0; i < N; i++) printf("%u ", a[i]); printf("\n");
        // for(int i = 0; i < N; i++) printf("%u ", b[i]); printf("\n");
        // for(int i = 0; i < N; i++) printf("%u ", array[i]); printf("\n");
    }
}

int main() {
    using std::chrono::high_resolution_clock;
    using std::chrono::duration;
    using std::chrono::milliseconds;

    int N = 1000000; // vector length

    uint *a = new uint[N];
    uint *b = new uint[N];
    uint *c = new uint[N];

    uchar key[] = { 0x00, 0x00, 0x00, 0x00,
                    0x00, 0x00, 0x00, 0x00,
                    0x00, 0x00, 0x00, 0x00,
                    0x00, 0x00, 0x00, 0x00 };
    uint keySize = 16;
    int Nr=10;
    uint e_sched[4*(MAXNR + 1)];
    uint d_sched[4*(MAXNR + 1)];
    makeKey(key, keySize << 3, DIR_BOTH, e_sched, d_sched, Nr);

    int runs = 100;
    double avg_time[3] = {0,0,0};
    for(int test=0; test < 3; test++){
        for(int i=0;i<runs;i++){

            // initiate
            for(int ii = 0; ii < N; ii++) {
                a[ii] = rand()%100;
                b[ii] = rand()%100;
            }

            auto t1 = high_resolution_clock::now();

            if(test==1){
                ltVectorAddition(c, a, b, N, e_sched, d_sched, Nr); // secure LITE's function
            }else if(test==0){
                uint *d_a, *d_b, *d_c;
                hipMalloc(&d_a, sizeof(uint)*N); hipMemcpy(d_a, a, sizeof(uint)*N, hipMemcpyHostToDevice);
                hipMalloc(&d_b, sizeof(uint)*N); hipMemcpy(d_b, b, sizeof(uint)*N, hipMemcpyHostToDevice);
                hipMalloc(&d_c, sizeof(uint)*N);
                vectorAdditionUnsecure<<<1024, 128>>>(d_c, d_a, d_b, N);      // unsecure
                hipMemcpy(c, d_c, sizeof(uint)*N, hipMemcpyDeviceToHost);
                hipFree(d_a); hipFree(d_b); hipFree(d_c);
            }else{
                for(int i=0;i<N;i++){
                    c[i] = a[i]+b[i];
                }
            }

            auto t2 = high_resolution_clock::now();
            
            duration<double, std::milli> ms_double = t2 - t1;
            avg_time[test] += ms_double.count();
            printf("%d  ", i); check(a, b, c, N);
            hipDeviceReset();
        }
        avg_time[test]/=runs;
    }
    cout << "GPU unsecure: " << avg_time[0]<< " ms\n"
         << "GPU secure  : " << avg_time[1] << " ms\n"
         << "CPU         : " << avg_time[2] << " ms\n";
}

// unsecure: 66.9532 ms
// secure  : 157.109 ms
