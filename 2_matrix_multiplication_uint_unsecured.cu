#include <iostream>
#include <hip/hip_runtime.h>
#include <chrono>
#include <vector>

#include "lite.cu"

#define TILE_SIZE 4

using namespace std;

void check(uint *a, uint *b, uint *res, int N){
    bool flag = false;
    uint *c = new uint[N*N];
    for(int i = 0; i < N; ++i) for(int j = 0; j < N; ++j) c[i*N+j] = 0;

    for(int i = 0; i < N; ++i)
        for(int j = 0; j < N; ++j)
            for(int k = 0; k < N; ++k){
                int row_a = i;
                int col_b = j;
                int cr_ab = k;
                c[row_a*N + col_b] += a[row_a*N + cr_ab]*b[cr_ab*N + col_b];
            }
    
    for(int i = 0; i < N; ++i)
        for(int j = 0; j < N; ++j)
            if(c[i*N+j]!=res[i*N+j]){
                flag = true;
            }
    if(!flag){
        cout << "SUCCESS" << endl;
    }else{
        cout << "FAIL" << endl;
    }
    // printf("ANS: ");
    // for(int i = 0; i < N; ++i)
    //     for(int j = 0; j < N; ++j)
    //         printf("%u ", c[i*N+j]);
    // printf("\nRES: ");
    // for(int i = 0; i < N; ++i)
    //     for(int j = 0; j < N; ++j)
    //         printf("%u ", res[i*N+j]);
    // printf("\n");
}

int main(){
    
    using chrono::high_resolution_clock;
    using chrono::duration;
    using chrono::milliseconds;

    int N = 200;  // Matrix size
    
    // Allocate host 
    uint *h_A = new uint[N * N];
    uint *h_B = new uint[N * N];
    uint *h_C = new uint[N * N];

    uchar key[] = { 0x00, 0x00, 0x00, 0x00,
                    0x00, 0x00, 0x00, 0x00,
                    0x00, 0x00, 0x00, 0x00,
                    0x00, 0x00, 0x00, 0x00 };
    uint keySize = 16;
    int Nr=10;
    uint e_sched[4*(MAXNR + 1)];
    uint d_sched[4*(MAXNR + 1)];
    makeKey(key, keySize << 3, DIR_BOTH, e_sched, d_sched, Nr);


    vector<double> times;
    for(int i = 0; i < 10; i++){ // benchmark run        
        // initialize
        for (int j = 0; j < N * N; ++j){
            h_A[j] = rand()%5+1;
            h_B[j] = rand()%10;
        }
        
        auto t1 = high_resolution_clock::now();
        liteMatMultiplication(h_C, h_A, h_B, N, e_sched, d_sched, Nr, true);
        auto t2 = high_resolution_clock::now();
        duration<double, std::milli> ms_double = t2 - t1;
        times.push_back(ms_double.count());
        
        check(h_A, h_B, h_C, N);   
        cout << "Time: " << ms_double.count() << " ms" << endl;
    }
    // average time
    double sum = 0;
    for(int i = 1; i < times.size(); i++){
        sum += times[i];
    }
    printf("Average time: %.3f ms\n", sum/(times.size()-1));
}