#include "hip/hip_runtime.h"
#define GRIDSIZE 256
#define BLOCKSIZE 128
#define BUFFSIZE 128 // to avoid splling data to global memory we only process 128 elements per each block

///////////////////////////////////////
// MAIN Lite's Vector-Vector Processing
///////////////////////////////////////
__global__ void vectorProc(uint *d_enc_result, uint *d_enc_a, uint *d_enc_b, int N,
                           uint *d_enc_sched, uint *d_dec_sched, int Nr, bool is_float,
                           uint procType=0){
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    
    if(index < N){
        __shared__ uint d_a[BUFFSIZE];
        __shared__ uint d_b[BUFFSIZE];
        __shared__ uint d_result[BUFFSIZE];
        
        __shared__ float d_f_a[BUFFSIZE];
        __shared__ float d_f_b[BUFFSIZE];
        __shared__ float d_f_result[BUFFSIZE];

        for(int idx = index; idx < N; idx += stride){
            // printf("%d %d %d %d %d %d\n", threadIdx.x, blockIdx.x, index, stride, idx, idx*4+3);

            // GPU Decrypt
            if(threadIdx.x%4==0){
                AES_decrypt_gpu(d_a + (idx % BUFFSIZE), d_enc_a + idx, d_dec_sched, Nr); 
                AES_decrypt_gpu(d_b + (idx % BUFFSIZE), d_enc_b + idx, d_dec_sched, Nr);  
            }

            __syncthreads();

            if(is_float){
                d_f_a[threadIdx.x] = *uintToFloat(&d_a[threadIdx.x]);
                d_f_b[threadIdx.x] = *uintToFloat(&d_b[threadIdx.x]);
                if(procType==0)
                    d_f_result[threadIdx.x] = d_f_a[threadIdx.x] + d_f_b[threadIdx.x];
                else if(procType==1)
                    d_f_result[threadIdx.x] = d_f_a[threadIdx.x] * d_f_b[threadIdx.x];
                else if(procType==2)
                    d_f_result[threadIdx.x] = d_f_a[threadIdx.x] - d_f_b[threadIdx.x];
                else if(procType==3)
                    d_f_result[threadIdx.x] = d_f_a[threadIdx.x] / d_f_b[threadIdx.x];
                d_result[threadIdx.x] = *floatToUint(&d_f_result[threadIdx.x]);
            }else{
                if(procType==0)
                    d_result[threadIdx.x] = d_a[threadIdx.x] + d_b[threadIdx.x];
                else if(procType==1)
                    d_result[threadIdx.x] = d_a[threadIdx.x] * d_b[threadIdx.x];
                else if(procType==2)
                    d_result[threadIdx.x] = d_a[threadIdx.x] - d_b[threadIdx.x];
                else if(procType==3)
                    d_result[threadIdx.x] = d_a[threadIdx.x] / d_b[threadIdx.x];
            }

            __syncthreads();
            
            // GPU Encrypt
            if(threadIdx.x%4==0){
                AES_encrypt_gpu(d_enc_result + idx, d_result + (idx % BUFFSIZE), d_enc_sched, Nr);
            }
        }
    }
}
// wrapper vector processing for CPU-GPU comm.
void ltVectorProc(uint *result, uint *a, uint *b, int N,
                      uint *enc_sched, uint *dec_sched, int Nr, bool is_float, int procType=0,
                      int gridSize=GRIDSIZE, int blockSize=BLOCKSIZE){
    // Check size, pad so it's divisible by 4
    int padSizeA = padArray(a, N);
    int padSizeB = padArray(b, N);
    
    N += padSizeA; // assuming the size is the same
    
    // CPU Encrypt N elements
    uint *enc_a = new uint[N];
    uint *enc_b = new uint[N];
    uint *enc_result = new uint[N];
    ltEncryptCPU(enc_a, a, enc_sched, Nr, N);
    ltEncryptCPU(enc_b, b, enc_sched, Nr, N);

    // CPU -> GPU: Data
    uint *d_enc_a, *d_enc_b, *d_enc_result;
    size_t size = sizeof(uint)*N;
    gpuErrchk( hipMalloc(&d_enc_a, size) );
    gpuErrchk( hipMalloc(&d_enc_b, size) );
    gpuErrchk( hipMalloc(&d_enc_result, size) );
    gpuErrchk( hipMemcpy(d_enc_a, enc_a, size, hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(d_enc_b, enc_b, size, hipMemcpyHostToDevice) );

    // CPU -> GPU: Key
    uint *d_enc_sched;
    uint *d_dec_sched;
    size_t key_size = (4*(MAXNR + 1)) * sizeof(uint);
    gpuErrchk( hipMalloc(&d_enc_sched, key_size) );
    gpuErrchk( hipMalloc(&d_dec_sched, key_size) );
    gpuErrchk( hipMemcpy(d_enc_sched, enc_sched, key_size, hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(d_dec_sched, dec_sched, key_size, hipMemcpyHostToDevice) );
    
    vectorProc<<<gridSize, blockSize>>>(d_enc_result, d_enc_a, d_enc_b, N,d_enc_sched, d_dec_sched, Nr, is_float, procType);

    // printf("----Leak Global Memory of The Result-------\n");
    // float *tmp = new float[N];
    // for(int i=0;i<N;i++) if(is_float) memcpy(&tmp[i], &result[i], sizeof(uint)),  printf("%.4f ", tmp[i]); else printf("%u ",result[i]); printf("\n");
    // printf("-------------------------------------------\n");

    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );

    // GPU -> CPU
    gpuErrchk( hipMemcpy(enc_result, d_enc_result, size, hipMemcpyDeviceToHost) );

    // CPU Decrypt
    ltDecryptCPU(result, enc_result, dec_sched, Nr, N);

    removePadArray(a, N, padSizeA);
    removePadArray(b, N, padSizeB);
    N -= padSizeA;
    
    hipFree(d_enc_a);
    hipFree(d_enc_b);
    hipFree(d_enc_sched);
    hipFree(d_dec_sched);
}

/////// ADDITION
// front-end wrapper vector addtion for uint array
void liteAddition(uint *result, uint *a, uint *b, int N,
                      uint *enc_sched, uint *dec_sched, int Nr,
                      int gridSize=GRIDSIZE, int blockSize=BLOCKSIZE){
   ltVectorProc(result, a, b, N, enc_sched, dec_sched, Nr, false, 0, gridSize, blockSize);
}
// front-end wrapper vector addtion for float array
void liteAddition(float *result, float *a, float *b, int N,
                      uint *enc_sched, uint *dec_sched, int Nr,
                      int gridSize=GRIDSIZE, int blockSize=BLOCKSIZE){
    // Float array to uint array
    uint *uint_a = new uint[N];
    uint *uint_b = new uint[N];
    uint *uint_result = new uint[N];
    floatToUintCPU(uint_a, a, N);
    floatToUintCPU(uint_b, b, N);

    ltVectorProc(uint_result, uint_a, uint_b, N, enc_sched, dec_sched, Nr, true, 0, gridSize, blockSize);

    // uint to float
    uintToFloatCPU(result, uint_result, N);
}

/////// MULTIPLICATION
// front-end wrapper vector multiplication for uint array
void liteMultiplication(uint *result, uint *a, uint *b, int N,
                      uint *enc_sched, uint *dec_sched, int Nr,
                      int gridSize=GRIDSIZE, int blockSize=BLOCKSIZE){
   ltVectorProc(result, a, b, N, enc_sched, dec_sched, Nr, false, 1, gridSize, blockSize);
}
// front-end wrapper vector multiplication for float array
void liteMultiplication(float *result, float *a, float *b, int N,
                      uint *enc_sched, uint *dec_sched, int Nr,
                      int gridSize=GRIDSIZE, int blockSize=BLOCKSIZE){
    // Float array to uint array
    uint *uint_a = new uint[N];
    uint *uint_b = new uint[N];
    uint *uint_result = new uint[N];
    floatToUintCPU(uint_a, a, N);
    floatToUintCPU(uint_b, b, N);

    ltVectorProc(uint_result, uint_a, uint_b, N, enc_sched, dec_sched, Nr, true, 1, gridSize, blockSize);

    // uint to float
    uintToFloatCPU(result, uint_result, N);
}
// front-end wrapper vector substraction for uint array
void liteSubstraction(uint *result, uint *a, uint *b, int N,
                      uint *enc_sched, uint *dec_sched, int Nr,
                      int gridSize=GRIDSIZE, int blockSize=BLOCKSIZE){
   ltVectorProc(result, a, b, N, enc_sched, dec_sched, Nr, false, 2, gridSize, blockSize);
}

/////// SUBSTRACTION
// front-end wrapper vector addtion for float array
void liteSubstraction(float *result, float *a, float *b, int N,
                      uint *enc_sched, uint *dec_sched, int Nr,
                      int gridSize=GRIDSIZE, int blockSize=BLOCKSIZE){
    // Float array to uint array
    uint *uint_a = new uint[N];
    uint *uint_b = new uint[N];
    uint *uint_result = new uint[N];
    floatToUintCPU(uint_a, a, N);
    floatToUintCPU(uint_b, b, N);

    ltVectorProc(uint_result, uint_a, uint_b, N, enc_sched, dec_sched, Nr, true, 2, gridSize, blockSize);

    // uint to float
    uintToFloatCPU(result, uint_result, N);
}

/////// DIVISION
// front-end wrapper vector division for uint array
void liteDivision(uint *result, uint *a, uint *b, int N,
                      uint *enc_sched, uint *dec_sched, int Nr,
                      int gridSize=GRIDSIZE, int blockSize=BLOCKSIZE){
   ltVectorProc(result, a, b, N, enc_sched, dec_sched, Nr, false, 3, gridSize, blockSize);
}
// front-end wrapper vector addtion for float array
void liteDivision(float *result, float *a, float *b, int N,
                      uint *enc_sched, uint *dec_sched, int Nr,
                      int gridSize=GRIDSIZE, int blockSize=BLOCKSIZE){
    // Float array to uint array
    uint *uint_a = new uint[N];
    uint *uint_b = new uint[N];
    uint *uint_result = new uint[N];
    floatToUintCPU(uint_a, a, N);
    floatToUintCPU(uint_b, b, N);

    ltVectorProc(uint_result, uint_a, uint_b, N, enc_sched, dec_sched, Nr, true, 3, gridSize, blockSize);

    // uint to float
    uintToFloatCPU(result, uint_result, N);
}