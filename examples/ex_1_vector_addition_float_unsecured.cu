/*
Example for float vector addition unsecure way
This code include the benchmarking code. See "benchmark run" in main() function.
*/
#include <stdio.h>
#include <iostream>
#include <typeinfo>
#include <vector>
#include <chrono>
#include "unsecure_lite.cu"

void check(float *a, float *b, float *array, int N){
    bool flag = false;
    for(int i = 0; i < N; i++){
        if(array[i] != a[i]+b[i]){
            flag = true;
        }
    }
    if(!flag){
        printf("SUCCESS\n");
        // for(int i = 0; i < N; i++) printf("%.3f ", a[i]); printf("\n");
        // for(int i = 0; i < N; i++) printf("%.3f ", b[i]); printf("\n");
        // for(int i = 0; i < N; i++) printf("%.3f ", array[i]); printf("\n");
    }else{
        printf("FAIL\n");
        // for(int i = 0; i < N; i++) printf("%.3f ", a[i]); printf("\n"); printf("\n");
        // for(int i = 0; i < N; i++) printf("%.3f ", b[i]); printf("\n"); printf("\n");
        // for(int i = 0; i < N; i++) printf("%.3f ", array[i]); printf("\n");
    }
}

int main() {
    
    using std::chrono::high_resolution_clock;
    using std::chrono::duration;
    using std::chrono::milliseconds;
    int N = 500; // vector length

    float *a = new float[N];
    float *b = new float[N];
    float *c = new float[N];


   vector<double> times;
    for(int i = 0; i < 50; i++){ // benchmark run
        // initiate
        for(int i = 0; i < N; i++) {
            a[i] = rand()%100 / 10.0;
            b[i] = rand()%100 / 10.0;
        }
        
        auto t1 = high_resolution_clock::now();
        
        ltVecAdd(c, a, b, N);
        
        auto t2 = high_resolution_clock::now();
        duration<double, std::milli> ms_double = t2 - t1;
        times.push_back(ms_double.count());
        
        check(a, b, c, N);
        hipDeviceReset();
    }
    // average time
    double sum = 0;
    for(int i = 0; i < times.size(); i++){
        sum += times[i];
    }
    printf("Average time: %.3f ms\n", sum/times.size());
}
