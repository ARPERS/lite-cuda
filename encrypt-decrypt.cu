#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <typeinfo>

#include "AES/AES_encrypt_cpu.cpp"
#include "AES/AES_encrypt_gpu.cu"
#include "AES/AES_decrypt_cpu.cpp"
#include "AES/AES_decrypt_gpu.cu"
#include "AES/AES.cu"

using namespace std;

__global__ void helloWorld(){
    printf("Halo from GPU!\n");
}

uint32_t floatToUInt(float value) {
  union {
    float floatValue;
    uint32_t uintValue;
  } u;

  u.floatValue = value;
  return u.uintValue;
}

float uintToFloat(uint32_t value) {
  union {
    float floatValue;
    uint32_t uintValue;
  } u;

  u.uintValue = value;
  return u.floatValue;
}

__global__ void AESEncryptGPU(uint *pt, const uint *ct, uint *rek, uint Nr){
      AES_encrypt_gpu(pt, ct, rek, Nr);
}

void AESEncryptCPU(uint *pt, const uint *ct, uint *rek, uint Nr){
    AES_encrypt_cpu(pt, ct, rek, Nr);
}

void AESDecryptCPU(uint *pt, const uint *ct, uint *rek, uint Nr){
      AES_decrypt_cpu(pt, ct, rek, Nr);
}

__global__ void AESDecryptGPU(uint *ct, const uint *pt, uint *rek, uint Nr){
      AES_decrypt_gpu(ct, pt, rek, Nr);
}

__global__ void setVal(uint *arr, int i, uint v){
    arr[i] = v;
}

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

int main() {
    int N = 4; // vector length

    // key declaration
    uchar key[] = { 0x00, 0x00, 0x00, 0x00,
                    0x00, 0x00, 0x00, 0x00,
                    0x00, 0x00, 0x00, 0x00,
                    0x00, 0x00, 0x00, 0x00 };
    uint keySize = 16;
    int Nr=10;
    uint e_sched[4*(MAXNR + 1)];
    uint d_sched[4*(MAXNR + 1)];
    makeKey(key, keySize << 3, DIR_BOTH, e_sched, d_sched, Nr);

    // Initiating values in CPU
    size_t bytes = N * sizeof(uint);
    uint *x = (uint*)malloc(bytes);
    uint *y = (uint*)malloc(bytes);
    uint *z = (uint*)malloc(bytes);
    x[0] = 123; x[1] = 222; x[2]=989; x[3]=275; 

    // Send Key to GPU
    uint *d_e_sched;
    uint *d_d_sched;
    size_t key_size = (4*(MAXNR + 1)) * sizeof(uint);
    gpuErrchk( hipMalloc(&d_e_sched, key_size) );
    gpuErrchk( hipMalloc(&d_d_sched, key_size) );
    gpuErrchk( hipMemcpy(d_e_sched, e_sched, key_size, hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(d_d_sched, d_sched, key_size, hipMemcpyHostToDevice) );

    // CPU
    cout << "CPU Pln Text: "<< x[0] << " " << x[1] << " " << x[2] << " " << x[3] << endl;
    AESEncryptCPU(y, x, e_sched, Nr);
    cout << "CPU Pln Text: "<< y[0] << " " << y[1] << " " << y[2] << " " << y[3] << endl;
    AESDecryptCPU(z, y, d_sched, Nr);
    cout << "CPU Pln Text: "<< z[0] << " " << z[1] << " " << z[2] << " " << z[3] << endl;
   
    cout << "-----------\n";

    // Initiating values in GPU
    uint *d_x, *d_y, *d_z;
    gpuErrchk( hipMalloc(&d_x, bytes) ); 
    gpuErrchk( hipMemcpy(d_x, x, bytes, hipMemcpyHostToDevice) );
    gpuErrchk( hipMalloc(&d_y, bytes) );
    gpuErrchk( hipMalloc(&d_z, bytes) );

    hipMemcpy(x, d_x, bytes, hipMemcpyDeviceToHost);
    cout << "GPU Pln Text: " << x[0] << " " << x[1] << " " << x[2] << " " << x[3] << endl;

    AESEncryptGPU<<< 1, 1 >>>(d_y, d_x, d_e_sched, Nr);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
   
    hipMemcpy(y, d_y, bytes, hipMemcpyDeviceToHost);
    cout << "GPU Chp Text: " << y[0] << " " << y[1] << " " << y[2] << " " << y[3] << endl;
    
    AESDecryptGPU<<<1,1>>>(d_z, d_y, d_d_sched, Nr);
    hipMemcpy(z, d_z, bytes, hipMemcpyDeviceToHost);
    cout << "GPU Pln Text: " << z[0] << " " << z[1] << " " << z[2] << " " << z[3] << endl;
    
    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_z);
}
