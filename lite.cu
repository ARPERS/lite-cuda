#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <typeinfo>
#include <hip/hip_runtime.h>

#include "AES/AES_encrypt_cpu.cpp"
#include "AES/AES_encrypt_gpu.cu"
#include "AES/AES_decrypt_cpu.cpp"
#include "AES/AES_decrypt_gpu.cu"
#include "AES/AES.cu"

using namespace std;

//0. Debugging
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true){
   if (code != hipSuccess){
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

//0. Utils
__device__ unsigned int* floatToUint(float  *input){
    unsigned char  * temp1 = reinterpret_cast<unsigned char  *>(input);       
    unsigned int  * output = reinterpret_cast<unsigned int  *>(temp1);    
    return output;
}
__device__ float * uintToFloat(unsigned int  *input){
    unsigned char  * temp1 = reinterpret_cast<unsigned char  *>(input);       
    float  * output = reinterpret_cast<float  *>(temp1);    
    return output;
}
void floatToUintCPU(uint *dest, const float *source, int N) {
    for(int i=0; i<N; i++) memcpy(&dest[i], &source[i], sizeof(float));
}
void uintToFloatCPU(float *dest, const uint *source, int N) {
    for(int i=0; i<N; i++) memcpy(&dest[i], &source[i], sizeof(uint));
}

//1. Encrypt-Decrypt Function
__device__ void ltEncryptGPU(uint *ct, const uint *pt, uint *rek, uint Nr){
    AES_encrypt_gpu(ct, pt, rek, Nr);
}

__device__ void ltDecryptGPU(uint *pt, const uint *ct, uint *rek, uint Nr){
    AES_decrypt_gpu(pt, ct, rek, Nr);
}

void ltEncryptCPU(uint *ct, const uint *pt, uint *rek, uint Nr){ // encrypt 4 elements pointer+0, +1, +2, +3
      AES_encrypt_cpu(ct, pt, rek, Nr);
}
void ltDecryptCPU(uint *pt, const uint *ct, uint *rek, uint Nr){ // encrypt 4 elements pointer+0, +1, +2, +3
      AES_decrypt_cpu(pt, ct, rek, Nr);
}
void ltEncryptCPU(uint *ct, const uint *pt, uint *rek, uint Nr, int N){ // run encrypt for all elements
    for(int i=0;i<N;i+=4){
      ltEncryptCPU(ct+i, pt+i, rek, Nr);
    }
}
void ltDecryptCPU(uint *pt, const uint *ct, uint *rek, uint Nr, int N){ // run encrypt for all elements
    for(int i=0;i<N;i+=4){
      ltDecryptCPU(pt+i, ct+i, rek, Nr);
    }
}

//2. MAIN Lite's Vector Addition
__global__ void vectorAddition(uint *d_enc_result, uint *d_enc_a, uint *d_enc_b, int N, uint *d_enc_sched, uint *d_dec_sched, int Nr, bool is_float){
    int index = threadIdx.x + blockIdx.x * blockDim.x;

    if(index * 4 < N){
        uint d_a[4], d_b[4];
        uint *d_result = new uint[4];

        // GPU Decrypt
        ltDecryptGPU(d_a, d_enc_a + index*4, d_dec_sched, Nr); 
        ltDecryptGPU(d_b, d_enc_b + index*4, d_dec_sched, Nr);  

        if(is_float){
            float *d_f_a = new float[4];
            float *d_f_b = new float[4];
            float *d_f_result =new float[4];
            d_f_a = uintToFloat(d_a);
            d_f_b = uintToFloat(d_b);
            for(int i = 0; i < 4; i ++){
                d_f_result[i] = d_f_a[i] + d_f_b[i];
            }
            d_result = floatToUint(d_f_result);
        }else{
            for(int i = 0; i < 4; i ++){
                d_result[i] = d_a[i] + d_b[i];
            }
        }
        // GPU Encrypt
        ltEncryptGPU(d_enc_result + index*4, d_result, d_enc_sched, Nr);
    }
}

// wrapper vector addtion CPU-GPU comm.
void ltVectorAddition(uint *result, uint *a, uint *b, int N, uint *enc_sched, uint *dec_sched, int Nr, bool is_float){
    // CPU Encrypt N elements
    uint *enc_a = new uint[N];
    uint *enc_b = new uint[N];
    uint *enc_result = new uint[N];
    ltEncryptCPU(enc_a, a, enc_sched, Nr, N);
    ltEncryptCPU(enc_b, b, enc_sched, Nr, N);

    // CPU -> GPU: Data
    uint *d_enc_a, *d_enc_b, *d_enc_result;
    size_t size = sizeof(uint)*N;
    gpuErrchk( hipMalloc(&d_enc_a, size) );
    gpuErrchk( hipMalloc(&d_enc_b, size) );
    gpuErrchk( hipMalloc(&d_enc_result, size) );
    gpuErrchk( hipMemcpy(d_enc_a, enc_a, size, hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(d_enc_b, enc_b, size, hipMemcpyHostToDevice) );

    // CPU -> GPU: Key
    uint *d_enc_sched;
    uint *d_dec_sched;
    size_t key_size = (4*(MAXNR + 1)) * sizeof(uint);
    gpuErrchk( hipMalloc(&d_enc_sched, key_size) );
    gpuErrchk( hipMalloc(&d_dec_sched, key_size) );
    gpuErrchk( hipMemcpy(d_enc_sched, enc_sched, key_size, hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(d_dec_sched, dec_sched, key_size, hipMemcpyHostToDevice) );
    
    vectorAddition<<<1, N/4>>>(d_enc_result, d_enc_a, d_enc_b, N, d_enc_sched, d_dec_sched, Nr, is_float);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );

    // GPU -> CPU
    gpuErrchk( hipMemcpy(enc_result, d_enc_result, size, hipMemcpyDeviceToHost) );

    // CPU Decrypt
    ltDecryptCPU(result, enc_result, dec_sched, Nr, N);
}

// wrapper vector addtion for uint array
void ltVectorAddition(uint *result, uint *a, uint *b, int N, uint *enc_sched, uint *dec_sched, int Nr){
   ltVectorAddition(result, a, b, N, enc_sched, dec_sched, Nr, false);
}

// wrapper vector addtion for float array
void ltVectorAddition(float *result, float *a, float *b, int N, uint *enc_sched, uint *dec_sched, int Nr){

    // debug
    // printf("BEFORE\n");
    // for(int i = 0; i < N; i++) printf("%f ", a[i]); printf("\n");

    // Float array to uint array
    uint *uint_a = new uint[N];
    uint *uint_b = new uint[N];
    uint *uint_result = new uint[N];
    floatToUintCPU(uint_a, a, N);
    floatToUintCPU(uint_b, b, N);
    floatToUintCPU(uint_result, result, N);

    // debug
    // printf("PUNNED to UINT\n");
    // for(int i = 0; i < N; i++) printf("%u ", uint_a[i]); printf("\n");

    ltVectorAddition(uint_result, uint_a, uint_b, N, enc_sched, dec_sched, Nr, true);

    // uint to float
    uintToFloatCPU(a, uint_a, N);
    uintToFloatCPU(b, uint_b, N);
    uintToFloatCPU(result, uint_result, N);
}


//3. LITE's Matrix Multiplication
// CUDA kernel for matrix multiplication
__global__ void matrixMultiplication(float *A, float *B, float *C, int N){
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    const int tile_size = 2;

    // Allocate shared memory for tiles
    __shared__ float As[tile_size][tile_size];
    __shared__ float Bs[tile_size][tile_size];

    float Cvalue = 0.0f;

    for (int k = 0; k < N / tile_size; ++k){
        // Load tiles into shared memory
        As[threadIdx.y][threadIdx.x] = A[row * N + (k * tile_size + threadIdx.x)];
        Bs[threadIdx.y][threadIdx.x] = B[(k * tile_size + threadIdx.y) * N + col];

        // Synchronize threads to ensure all data is loaded
        __syncthreads();

        // Perform tile-wise matrix multiplication
        for (int i = 0; i < tile_size; ++i){
            Cvalue += As[threadIdx.y][i] * Bs[i][threadIdx.x];
        }

        // Synchronize threads to ensure all data is used before loading the next tiles
        __syncthreads();
    }

    // Store the result in global memory
    C[row * N + col] = Cvalue;
}